
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime_api.h>

#define SQ(x) ((x) * (x))

static const float A = -4.0, B = 4.0;  // limites de integración
static const int N = 1 << 22;          // número de intervalos = 2^22
static const float H = (B - A) / N;    // tamaño del intervalo de integración
static const float PI(M_PI);           // π con precision simple

__device__ float h(float x) {
    return .5f + 1.5f / (1.0f + 50.0f * SQ(x));
}

float host_h(float x) {
    return .5f + 1.5f / (1.0f + 50.0f * SQ(x));
}

__device__ float f(float x) {
    int i;
    float sum = 0.0f, x0;
    for (i = 0; i < 10; ++i){
        x0 = -3.3f + i * 0.7f;
        sum += h(x - x0);
    }
    return sum/10.0f;
}

float host_f(float x) {
    int i;
    float sum = 0.0f, x0;
    for (i = 0; i < 10; ++i){
        x0 = -3.3f + i * 0.7f;
        sum += host_h(x - x0);
   }
    return sum/10.0f;
}

__device__ float g(float x) {
    float c = cosf(2.0f * PI * f(x) * x);
    return expf(-x/16.0f) * SQ(c);
}


float host_g(float x) {
    float c = cosf(2.0f * PI * host_f(x) * x);
    return expf(-x/16.0f) * SQ(c);
}

__global__ void
integrate_blocks(float subtotals[]) {

	// Inicializar variable __shared__ 
	extern __shared__ float partialValues[];

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int tx = threadIdx.x;

	// Cada thread según su id, calcula la función de g(x)
	float x = 0; 

	// Realizamos el cálcul
	x = A + i * (H / 2);
	partialValues[tx] += (i % 2 == 0 ? 4 : 2) * g(x);


	 // Sincronizamos las hebras una vez que termine
	__syncthreads(); 

	// Hacer una suma por reduccion con los valores del arreglo subtotals[]
	for (int offset = blockDim.x/2; offset > 0; offset >>=1){
	        if (tx < offset)
	        {
	            //Añadimos una suma parcial con el offset
	            partialValues[tx] += partialValues[tx+offset];
	        }
	        __syncthreads();
	}
        // Escribimos el resultado del primer elemento de nuestro arreglo
        if(tx == 0)
        {
        // Resultados lo guarda por bloques y no por hebras
        subtotals[blockIdx.x] = partialValues[0];
        }
}


int main(int argc, char *argv[]) {

    // El programa recibe como parámetro el número de hebras por bloque.
    // Recuerden que este número debe ser múltiplo de 32 (tamaño del warp)
    // y puede ser a lo más 512 (limitación del hardware).

    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " threads_per_block" << std::endl;
        std::exit(1);
    }
    int block_size = std::atoi(argv[1]);

    // Al usar N subintervalos, hay que evaluar la función en 2N + 1 puntos.
    // Para paralelizar, mejor evaluar 2N puntos y sumar el último al final.
    // Por lo tanto, necesitamos 2N hebras.
    int nr_blocks = 2 * N / block_size;

    size_t sharedMem = 512 ;

    // Reservar arreglos en RAM y en la GPU para guardar los resultados.
    float *subtotals_h, *subtotals_d;
    subtotals_h = new float[nr_blocks];
    hipMalloc((void **) &subtotals_d, sizeof(float) * nr_blocks);

    integrate_blocks<<<nr_blocks, block_size,sharedMem>>>(subtotals_d);

    // En la parte (a) de la tarea,
    // la reducción global la hacemos en la CPU.

    hipMemcpy(subtotals_h, subtotals_d,
               sizeof(float) * nr_blocks, hipMemcpyDeviceToHost);

    float sum = 0.0;
#   pragma omp parallel for reduction(+: sum)
    for (int b = 0; b < nr_blocks; ++b) {
        sum += subtotals_h[b];
    }
    
    sum+= host_g(B) + host_g(A);

    float integral = sum * H / 6.0f;

    std::cout << "Integral: " << std::setprecision(5) << integral << std::endl;
    sum = 0.0;
    hipFree(subtotals_d);
    std::free(subtotals_h);
}


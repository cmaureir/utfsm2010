
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <fstream>
using namespace std;
using std::ifstream;
#define BLOCK_SIZE  2

// max 40
// 32
// 25
// 20
// 16
// 10
// 8
// 4
// min 2

// Device multiplication function called by Mul()
// Compute C = A * B
//	wA is the width of A
//	wB is the width of B

float A[1600][1600],B[1600][1600];

__global__ void Muld(float* A, float* B, int wA, int wB, float* C)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index of the first sub-matrix of A processed by the block
	int aBegin = wA * BLOCK_SIZE * by;

	// Index of the last sub-matrix of A processed by the block
	int aEnd = aBegin + wA - 1;

	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;

	// Step size used to iterate through the sub-matrices of B
	int bStep = BLOCK_SIZE * wB;

	// The element of the block sub-matrix that is computed by the thread
	float Csub = 0;

	// Loop	over all the sub-matrices of A and B required to compute the block
	for (int a = aBegin, b = bBegin; 
		a <= aEnd;
		a += aStep, b += bStep) {

		// Shared memory for the sub-matrix of A
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

		// Shared memory for the sub-matrix of B
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from global memory to shared memory;
		// each thread loads one element of each matrix
		As[ty][tx] = A[a + wA * ty + tx];
		Bs[ty][tx] = B[b + wB * ty + tx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together;
		// each thread computes one element
		// of the block sub-matrix
		for (int k = 0; k < BLOCK_SIZE; ++k)
			Csub += As[ty][k] * Bs[k][tx];

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to global memory;
	// each thread writes one element

	int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	C[c + wB * ty + tx] = Csub;
}
// Host multiplication function
// Compute C = A * B
// 	hA is the height of A
// 	wA is the width of A
//	wB is the width of B

void Mul(const float* A, const float* B, int hA, int wA, int wB,float* C)
{
	int size;
	// Load A and B to the device
	float* Ad;
	size = hA * wA * sizeof(float);
	hipMalloc((void**)&Ad, size);
	hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);

	float* Bd;
	size = wA * wB * sizeof(float);
	hipMalloc((void**)&Bd, size);
	hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

	// Allocate C on the device
	float* Cd;
	size = hA * wB * sizeof(float);
	hipMalloc((void**)&Cd, size);

	// Compute the execution configuration assuming
	// the matrix dimensions are multiples of BLOCK_SIZE
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(wB / dimBlock.x, hA / dimBlock.y);

	// Launch the device computation
	Muld<<<dimGrid, dimBlock>>>(Ad, Bd, wA, wB, Cd);

	//cout << cudaGetErrorString(cudaGetLastError()) << endl;

	// Read C from the device
	hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);


	// Free device memory
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);
}

void readFile(){
    int i=0,j=0,tmp=0;
    ifstream f;
    int num;
    bool t = true;
    f.open("1");
    f >> num;
    tmp += 1;
    while(!f.eof())
     {
        if(j == 1600){
            j=0;
            i++;
            //cout << endl;
            //getchar();
        }
        if(i == 1600){
            t = false;
            i = 0;
            j = 0;
        }
        if(t){
            A[i][j] = num;
        }
        else{
            B[i][j] = num;
        }
        f >> num;
        j++;
     }
    f.close();
}

int main(int argc , char* argv[]){
    timespec ts, te;
 
    float *C;
    C = (float*)malloc(sizeof(float) * 1600 * 1600);
	
	readFile();
    clock_gettime(CLOCK_REALTIME, &ts);
    Mul(*A,*B,1600,1600,1600,C);
    clock_gettime(CLOCK_REALTIME, &te);
    cout  << BLOCK_SIZE << " " <<  (te.tv_sec-ts.tv_sec)<<"."<<abs(te.tv_nsec-ts.tv_nsec) << endl;

	free(C);
    return 0;
}


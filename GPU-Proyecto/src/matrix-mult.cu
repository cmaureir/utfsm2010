// Thread block size:

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h> 
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <fstream>
using std::ifstream;
#define BLOCK_SIZE 16

using namespace std;

// Device multiplication function called by Mul()
// Compute C = A * B
//	wA is the width of A
//	wB is the width of B

float A[1600][1600],B[1600][1600];

__global__ void Muld(float* A, float* B, int wA, int wB, float* C)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index of the first sub-matrix of A processed by the block
	int aBegin = wA * BLOCK_SIZE * by;

	// Index of the last sub-matrix of A processed by the block
	int aEnd = aBegin + wA - 1;

	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;

	// Step size used to iterate through the sub-matrices of B
	int bStep = BLOCK_SIZE * wB;

	// The element of the block sub-matrix that is computed by the thread
	float Csub = 0;

	// Loop	over all the sub-matrices of A and B required to compute the block
	for (int a = aBegin, b = bBegin; 
		a <= aEnd;
		a += aStep, b += bStep) {

		// Shared memory for the sub-matrix of A
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

		// Shared memory for the sub-matrix of B
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from global memory to shared memory;
		// each thread loads one element of each matrix
		As[ty][tx] = A[a + wA * ty + tx];
		Bs[ty][tx] = B[b + wB * ty + tx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together;
		// each thread computes one element
		// of the block sub-matrix
		for (int k = 0; k < BLOCK_SIZE; ++k)
			Csub += As[ty][k] * Bs[k][tx];

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to global memory;
	// each thread writes one element

	int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	C[c + wB * ty + tx] = Csub;
}
// Host multiplication function
// Compute C = A * B
// 	hA is the height of A
// 	wA is the width of A
//	wB is the width of B

void Mul(const float* A, const float* B, int hA, int wA, int wB,float* C)
{
	int size;
	// Load A and B to the device
	float* Ad;
	size = hA * wA * sizeof(float);
	hipMalloc((void**)&Ad, size);
	hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);

	float* Bd;
	size = wA * wB * sizeof(float);
	hipMalloc((void**)&Bd, size);
	hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

	// Allocate C on the device
	float* Cd;
	size = hA * wB * sizeof(float);
	hipMalloc((void**)&Cd, size);

	// Compute the execution configuration assuming
	// the matrix dimensions are multiples of BLOCK_SIZE
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(wB / dimBlock.x, hA / dimBlock.y);

	// Launch the device computation
	Muld<<<dimGrid, dimBlock>>>(Ad, Bd, wA, wB, Cd);

	cout << hipGetErrorString(hipGetLastError()) << endl;

	// Read C from the device
	hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);


	// Free device memory
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);

	cout << "End Mul" << endl;
}

void readFile(){
    int i=0,j=0,tmp=0;
    ifstream f;
    int num;
    bool t = true;
    f.open("1");
    f >> num;
    tmp += 1;
    while(!f.eof())
     {
        if(j == 1600){
            j=0;
            i++;
            //cout << endl;
            //getchar();
        }
        if(i == 1600){
            t = false;
            i = 0;
            j = 0;
        }
        if(t){
            A[i][j] = num;
        }
        else{
            B[i][j] = num;
        }
        f >> num;
        j++;
     }
    f.close();
}




//Main function.
int main(int argc , char* argv[]){
   
   // float A[3][3] = { {2,3,4}, {1,3,5}, {2,8,7}};
   // float B[3][3] = { {2,3,4}, {1,3,5}, {2,8,7}};
    int i;
    float *C;
	//C = new float *[3];
    C = (float*)malloc(sizeof(float) * 1600 * 1600);
//
//    for(i = 0; i < 9; i++);
//    {
//        C[i] = (float*)malloc(3 * sizeof(float));
////        memset(C[i], 0, 3 * sizeof(float));
//    }
//
//
////    //Call CPU version

	readFile();

    Mul(*A,*B,1600,1600,1600,C);
	cout << "----" << endl;
	for(i=0;i<1600;i++){
			cout << C[i] << " ";
	}
	cout << endl;
	cout << "----" << endl;
//    //free memory from C
//    for(i = 0; i <= 9; i++);
//    {
//        free(C[i]);
//    }
	free(C);
    return 0;
}//end of main().


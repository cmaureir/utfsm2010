#include "hip/hip_runtime.h"
#include "gpu-map.hpp"
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdio.h>


// El kernel d_f implementa la función f aplicadas a un valor x cualquiera.
// Este kernel está declarado con el calificador __device__,
// por lo que sólo puede ser llamado por código que está siendo ejecutado
// en la GPU.

__device__ float d_f(float x, int M) {
    float s = 0.0;
    // 10000
    for (int k = 1; k <= M; ++k) {
        s += sinf(2 * float(M_PI) * k * x);
    }
    return s;
}

// El kernel map_f se encarga de que cada hebra
// ejecute la función d_f con el argumento apropiado.

__global__ void map_f(float x[], int M) {
    unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
    x[i] = d_f(x[i],M);
}

// gpu_map debe hacer toda la burocracia de ejecutar código en la GPU:
// reserva memoria, copia el arreglo, lanza el kernel,
// copia el resultado de vuelta y libera la memoria.

void gpu_map(float x[], unsigned n, int M) {

    int threadsPerBlock = 256;
    int blocksPerGrid = n/threadsPerBlock;

    // Variables para manejar los eventos de inicio y fin    
    hipEvent_t start, stop;

    // Nueva variable para el tiempo en la GPU
    float gpuTime;

    // Nueva variable para el calculo en la GPU
    float *d;

    // Reserva de memoria en GPU
    hipMalloc( (void**) &d, n*sizeof(float));

    // Copiar el arreglo a la memoria de la GPU
    hipMemcpy(d, x, n*sizeof(float), hipMemcpyHostToDevice);

    // Cramos los dos eventos de inicio y fin de procesamiento
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Establecemos el inicio del procesamiento
    hipEventRecord(start, 0); 

    // Llamada del kernel
    map_f<<<blocksPerGrid, threadsPerBlock>>> (d,M);

    // Sincronizamos las hebras una vez que termine el llamado del kernel
    hipDeviceSynchronize();

    // Establecemos el fin del procesamiento
    hipEventRecord(stop, 0);
    
    // Sincronizamos la detencion del proceso
    hipEventSynchronize(stop);

    // Obtenemos el tiempo que demoró el proceso en milisegundos
    hipEventElapsedTime(&gpuTime, start, stop);

    // Eliminamos ambos eventos
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copiar resultado de vuelta
    hipMemcpy(x,d, n * sizeof(float), hipMemcpyDeviceToHost);

    // Liberación de memoria
    hipFree(d);

	// Comentar para tests
    printf("Tiempo GPU: %f\n",gpuTime/1000);
	
	// Descomentar para tests
    // printf("%f ",gpuTime/1000);

}

